#include <hip/hip_runtime.h>
#include <stdio.h>

#define FULL_MASK 0xffffffff
#define __WARP_SIZE__ 32

__global__
void dot_product(double *data, int N, double* result) {
  size_t i = threadIdx.x + blockIdx.x*blockDim.x;
  double sum = 0.0;
  if (i < N)
    sum = data[i]*data[i];

  __syncwarp() ; //sync lanes within warp
  for (int offset = __WARP_SIZE__/2; offset > 0; offset /= 2)
    sum += __shfl_down_sync(FULL_MASK, sum, offset);

  __shared__ double s_mem[1024/__WARP_SIZE__];

  int nwarps = blockDim.x/__WARP_SIZE__;
  int warpId = threadIdx.x/__WARP_SIZE__;

  if (threadIdx.x % __WARP_SIZE__ == 0) {
    s_mem[warpId] = sum;
    printf("smem = %f\n", s_mem[warpId]);
  }

  __syncthreads(); //sync threads within block
  if (threadIdx.x == 0) {
    printf("nwarps = %d\n", nwarps);
    for (int j = 0; j < nwarps; ++j) {
      printf("value = %f, smem = %f\n", result[0], s_mem[j]);
      result[blockIdx.x] += s_mem[j];
    }
      
  }
}

double sum_arr(double *result, int numResults) {
  double sum = 0.0;
  for (int i = 0; i < numResults; ++i)
    sum += result[i];
  return sum;
}

int main(){
  int N = 2000;
  double *X;
  

  X = new double[N];
  for (auto i = 0; i < N; i ++ ) 
    X[i] = 1;

  double *x_d;
  double *result_d, *result_h;

  hipMalloc(&x_d, N*sizeof(double));
  hipMemcpy(x_d, X, N*sizeof(double), hipMemcpyHostToDevice);
  
  dim3 nthreads(256,1,1);
  dim3 nblocks( (N+nthreads.x-1)/nthreads.x,1,1);
  
  hipHostAlloc((void **)&result_h, nblocks.x*sizeof(double), hipHostMallocMapped);
  hipHostGetDevicePointer((void **)&result_d, (void *)result_h , 0);
  // result_h = new double[nblocks.x];

  dot_product<<<nblocks,nthreads,0,0>>>(x_d, N, result_d);
  hipDeviceSynchronize();
  // cudaMemcpy(result_h, result_d, nblocks.x*sizeof(double), cudaMemcpyDeviceToHost);
  hipFree(x_d);

  double value = sum_arr(result_h, nblocks.x);
  printf("value = %f\n", value);

  delete[] X;
}